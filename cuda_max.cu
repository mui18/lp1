
#include <hip/hip_runtime.h>
#include<stdio.h>
#define SIZE 100

__global__
	void max(int *a,int *b,int *c)
	{
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		int id;
		
		switch(i)
		{
			case 0:
				
				for(id = 0 ; id<SIZE/10 ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			break;			
			case 1:
			
			
				for(id = SIZE/10 ; id<(SIZE/10)+10 ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			
			break;
			case 2:
			
			
				for(id = (SIZE/10)+10 ; id< (SIZE/10)+20 ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			
			break;
			case 3:
			
			
				for(id = (SIZE/10)+20 ; id< (SIZE/10)+30 ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			
			break;
			case 4:
			
				
				for(id = (SIZE/10)+30 ; id<(SIZE/10)+40  ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			
			break;
			case 5:
			
			
				for(id = (SIZE/10)+40 ; id<(SIZE/10)+50  ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			break;
			case 6:
			
			
				for(id = (SIZE/10)+50 ; id< (SIZE/10)+60 ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			break;
			case 7:
			
				
				for(id =(SIZE/10)+60 ; id< (SIZE/10)+70 ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			break;
			case 8:
			
				for(id =(SIZE/10)+70 ; id< (SIZE/10)+80 ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			
			break;
			case 9:
					
				for(id =(SIZE/10)+80 ; id<(SIZE/10)+90  ; id++)
					{
						if(a[id] >	c[i])
							c[i] = a[id];
							
					}
				b[i] = c[i];
			
			break;
			
			}
		
	}
	
	__global__
	void max2(int *a , int *b , int *c)
	{
		
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int big=-9999,big2 = -9999;
	int id;
	

	if ( i==0 )
	{
	
		
		for(id =0 ; id<5 ; id++)
					{
		
						if(a[id] >	big)
							big = a[id];
							
					}
				*b = big;
	
		
		
	}
		
	if(i==1)
	{
		for(id =5 ; id<10 ; id++)
					{
						if(a[id] >	big2)
							big2 = a[id];
							
					}
				*c = big2;
		
		
		
	}
		
		
	}



int main()
{
	int a[100],b=0,c=0,i,big[10],big2[10];
	int *d_a,*d_b,*d_c,*d_d,*d_one,*d_two;
	
	
	for(i=0;i<SIZE;i++)
		{
			a[i] = rand();
			
			}	
		
		
			for(i=0;i<SIZE/10;i++)
			{
				big[i] = 0;
				big2[i] = -9999;
			
			}	
			
	printf("\n\tPrinting Array : ");		
	for(i=0;i<SIZE;i++)
		printf("\n%d",a[i]);
		
	hipMalloc(&d_a,SIZE*sizeof(int));
	hipMemcpy(d_a,a,SIZE*sizeof(int),hipMemcpyHostToDevice);
	
	hipMalloc(&d_b,SIZE*sizeof(int));
	hipMemcpy(d_b,big,SIZE*sizeof(int),hipMemcpyHostToDevice);
	
	hipMalloc(&d_c,SIZE*sizeof(int));
	hipMemcpy(d_c,big2,SIZE*sizeof(int),hipMemcpyHostToDevice);
	
	max<<<1,10>>>(d_a,d_b,d_c);
	hipDeviceSynchronize();
	
	hipMemcpy(big,d_b,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	
	printf("\n\n\tPrinting Max elements among array of 100 : ");
	
	for(i=0;i<SIZE/10;i++)
		printf("\n\t%d",big[i]);
	
	hipMalloc(&d_d,10*sizeof(int));
	hipMemcpy(d_d,big,10*sizeof(int),hipMemcpyHostToDevice);
	
	hipMalloc((void**)&d_one,sizeof(int));
	hipMalloc((void**)&d_two,sizeof(int));
	
	
	max2<<<1,2>>>(d_d,d_one,d_two);
	hipDeviceSynchronize();
	
	hipMemcpy(&b,d_one,sizeof(int),hipMemcpyDeviceToHost);
	
	
	hipMemcpy(&c,d_two,sizeof(int),hipMemcpyDeviceToHost);
	
	
	if(b>c)
		printf("\n\tMax = %d",b);
	else
		printf("\n\tMax = %d",c);
				
}