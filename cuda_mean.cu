
#include <hip/hip_runtime.h>
#include<stdio.h>


__global__
	void mean(int *a,int *b)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x;
		
		b[id] += a[id];
	
	}

int main()
{
	int a[100],b[100];
	int i,sum=0;
	int *dev_a,*dev_b;
	
	for(i=0;i<100;i++)
	{
		a[i] = 1;
		b[i] = 1;
		}
	
	
	printf("\n\t Printing Arrays : ");
	
	printf("Array A");
	for(i=0;i<100;i++)
	{
		printf("\n\t %d" ,a[i]);
	}

	printf("Array B");
	for(i=0;i<100;i++)
	{
		printf("\n\t %d" ,b[i]);
	}
	
	hipMalloc(&dev_a,100*sizeof(int));
	hipMalloc(&dev_b,100*sizeof(int));

	hipMemcpy(dev_a,a,100*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,100*sizeof(int),hipMemcpyHostToDevice);
	
	mean<<<1,100>>>(dev_a,dev_b);
	
	hipMemcpy(&b,dev_b,100*sizeof(int),hipMemcpyDeviceToHost);
	
	
	for(i=0;i<100;i++)
	{
		
		sum+=b[i];
	}
	
	printf("\n\tSum = %d",sum);
	
	printf("\n\tMean = %d",sum/100);
	
}