#include "hip/hip_runtime.h"
    #include <hip/hip_runtime.h>
    #include <stdio.h>
    #include <time.h>

    #define SIZE 10

    __global__ void min(int *a , int *c,int *d)	// kernel function definition
    {
    int i = blockIdx.x * blockDim.x + threadIdx.x;					// initialize i to thread ID
	printf("Thread number= %d",i);
	if(i==0)
	{
	int small=9999;
		int j;
		for(j=0;j<5;j++)
			{

				if(a[j]<small){
					small = a[j];
					}
			}
								*c = small;
					printf("\n\tsmall1 = %d small2 = %d",small,small2);

	}
	
	if(i==1)
	{
		int small2=9999;
			int j;
		for(j=5;j<10;j++)
			{
				
				if(a[j]<small2){
					small2 = a[j];
					}
			}
					*d = small2;
					printf("\n\tsmall1 = %d small2 = %d",small,small2);

	
	}
		

    }

    int main()
    {
    int i;
    srand(time(NULL));		//makes use of the computer's internal clock to control the choice of the seed

    int a[SIZE] = {10,3,6,2,9,1,0,8,5,8};
    int c,d;

    int *dev_a, *dev_c,*dev_d;		//GPU / device parameters

    hipMalloc((void **) &dev_a, SIZE*sizeof(int));		//assign memory to parameters on GPU from CUDA runtime API
    hipMalloc((void **) &dev_c, SIZE*sizeof(int));
    hipMalloc((void **) &dev_d, SIZE*sizeof(int));

    

    /*for( i = 0 ; i < SIZE ; i++)
    {
    	a[i] = j;
    	j--;		// input the numbers
    }*/
    for( i = 0 ; i < SIZE ; i++)
    {
    	printf("\n\t%d", a[i]);			// input the numbers
    }
    
    hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);		//copy the array from CPU to GPU
    min<<<1,2>>>(dev_a,dev_c,dev_d);										// call kernel function <<<number of blocks, number of threads
    hipMemcpy(&c, dev_c, SIZE*sizeof(int),hipMemcpyDeviceToHost);		// copy the result back from GPU to CPU
	hipMemcpy(&d, dev_d, SIZE*sizeof(int),hipMemcpyDeviceToHost);

	if(c>d)
	    printf("\nmin =  %d ",d);		
	else
		    printf("\nmin =  %d ",c);


    hipFree(dev_a);		// Free the allocated memory
    hipFree(dev_c);
    printf("");

    return 0;
    }
