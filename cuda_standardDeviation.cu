
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__
void sub(float *a,float *b,int count,float mean)
{
	int id = blockIdx.x  *  blockDim.x + threadIdx.x;
	
	if(id<count)
	{
		
		b[id] = (a[id]-mean)*(a[id]-mean);
	}
	
}


int main(void)
{
float h_a[10],h_b[10],mean,std_dev=0,var=0;
int i,count=10,sum=0;
float *d_a,*d_b;

for(i=0;i<count;i++)
{
h_a[i] = i;
h_b[i] = 0.0;

}
printf("\n\tPrinting Array: ");

for(i=0;i<count;i++)
{
	printf("\n\t %f  ",h_a[i]);
}



printf("\n\n\tAddition of Array = ");
for(i=0;i<count;i++)
{
	sum+=h_a[i];
}

printf(" %d",sum);

mean = (float)sum/count;

printf("\n\tMean = %f",mean);


hipMalloc(&d_a,sizeof(int)*count);
hipMemcpy(d_a,h_a,sizeof(int)*count,hipMemcpyHostToDevice);

hipMalloc(&d_b,sizeof(int)*count);
hipMemcpy(d_b,h_b,sizeof(int)*count,hipMemcpyHostToDevice);


sub<<<1,10>>>(d_a,d_b,count,mean);

hipMemcpy(h_b,d_b,sizeof(int)*count,hipMemcpyDeviceToHost);



for(i=0;i<count;i++)
{
	var+=h_b[i];
}


var = var/(count);

printf("\n\tVariance = %f",var);

std_dev = sqrt(var);

printf("\n\tStandard Deviation = %f",std_dev);

hipFree(d_a);
hipFree(d_b);
return 0;

}

